#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

extern "C" {
    #include "sw_solve_gpu_sm.h"

    #include "sw_prefs.h"
    #include "sw_data.h"
    #include "chain.h"
    #include "matcher.h"
    #include "utils.h"
}

#define ALPHA 5
#define GPU_CNST CONF_SLV_SM

#include "templates/sm.template.h"
#include "templates/generic.h"

#include "templates/sw_solve.template"

extern SWData* swSolveGPUSM(Chain* rowChain, Chain* columnChain, 
    SWPrefs* swPrefs) {

    SWData* swData = swSolveGPU(rowChain, columnChain, swPrefs);

    return swData;
}
