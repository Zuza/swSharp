#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

extern "C" {
    #include "nw_solve_gpu_mm.h"

    #include "sw_prefs.h"
    #include "sw_data.h"
    #include "chain.h"
    #include "matcher.h"
    #include "utils.h"
}

#define ALPHA 5
#define GPU_CNST NW_CONF_SLV_MM

#include "templates/mm.template.h"
#include "templates/generic.h"

#include "templates/nw_solve.template"

extern SWData* nwSolveGPUMM(Chain* rowChain, Chain* columnChain, 
    SWPrefs* swPrefs) {

    SWData* swData = nwSolveGPU(rowChain, columnChain, swPrefs);

    return swData;
}
