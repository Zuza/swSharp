/*
SW# - CUDA parallelized Smith Waterman with applying Hirschberg's algorithm
Copyright (C) 2011 Matija Korpar

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

Contact the author by mkorpar@gmail.com.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

extern "C" {
    #include "nw_solve_gpu_mm.h"

    #include "sw_prefs.h"
    #include "sw_data.h"
    #include "chain.h"
    #include "matcher.h"
    #include "utils.h"
}

#define ALPHA 5
#define GPU_CNST NW_CONF_SLV_MM

#include "templates/mm.template.h"
#include "templates/generic.h"

#include "templates/nw_solve.template"

extern SWData* nwSolveGPUMM(Chain* rowChain, Chain* columnChain, 
    SWPrefs* swPrefs) {

    SWData* swData = nwSolveGPU(rowChain, columnChain, swPrefs);

    return swData;
}
