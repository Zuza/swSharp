#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

extern "C" {
    #include "sw_find_gpu_mm.h"

    #include "sw_prefs.h"
    #include "sw_data.h"
    #include "chain.h"
    #include "matcher.h"
    #include "utils.h"
}

#define ALPHA 5
#define SCORE CONF_FIND_SCORE_SS
#define GPU_CNST CONF_FIND_MM

#include "templates/mm.template.h"
#include "templates/generic.h"

#include "templates/sw_find.template"

extern SWData* swFindGPUMM(Chain* rowChain, Chain* columnChain, 
    SWPrefs* swPrefs, MatcherScore score) {

    SWData* swData = swFindGPU(rowChain, columnChain, swPrefs, score);

    return swData;
}
