/*
SW# - CUDA parallelized Smith Waterman with applying Hirschberg's algorithm
Copyright (C) 2011 Matija Korpar

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

Contact the author by mkorpar@gmail.com.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

extern "C" {
    #include "sw_solve_shotgun_gpu_mm.h"

    #include "sw_prefs.h"
    #include "sw_data.h"
    #include "chain.h"
    #include "matcher.h"
    #include "utils.h"
}

#define ALPHA 5
#define GPU_CNST SHOTGUN_CONF_SLV_MM

#include "templates/mm.template.h"
#include "templates/generic.h"

#include "templates/sw_solve_shotgun.template"

extern SWData* swSolveShotgunGPUMM(Chain* rowChain, Chain* columnChain, 
    SWPrefs* swPrefs) {

    SWData* swData = swSolveShotgunGPU(rowChain, columnChain, swPrefs);

    return swData;
}
