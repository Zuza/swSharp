/*
SW# - CUDA parallelized Smith Waterman with applying Hirschberg's algorithm
Copyright (C) 2011 Matija Korpar

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

Contact the author by mkorpar@gmail.com.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <limits.h>

extern "C" {
    #include "sw_solve_gpu.h"
    #include "sw_reconstruct_gpu_mm.h"

    #include "sw_hirschberg_data.h"
    #include "sw_prefs.h"
    #include "sw_data.h"
    #include "chain.h"
    #include "matcher.h"
    #include "utils.h"
}

#define ALPHA 1
#define GPU_CNST CONF_RCN_MM

#include "templates/mm.template.h"
#include "templates/generic.h"

#include "templates/sw_reconstruct.template"

extern SWHirschbergData* swReconstructGPUMM(Chain* rowChain, Chain* columnChain, 
    SWPrefs* swPrefs) {

    int rows = chainGetLength(rowChain);
    int columns = chainGetLength(columnChain);

    SWHirschbergData* data = swHirschbergDataCreate();
    HirshbergParent parent = {0, 0, 0, 0, rows - 1, columns - 1};

    hirschberg(data, rowChain, columnChain, swPrefs, parent);

    return data;
}
