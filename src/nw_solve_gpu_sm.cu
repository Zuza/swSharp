#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

extern "C" {
    #include "nw_solve_gpu_sm.h"

    #include "sw_prefs.h"
    #include "sw_data.h"
    #include "chain.h"
    #include "matcher.h"
    #include "utils.h"
}

#define ALPHA 5
#define GPU_CNST NW_CONF_SLV_SM

#include "templates/sm.template.h"
#include "templates/generic.h"

#include "templates/nw_solve.template"

extern SWData* nwSolveGPUSM(Chain* rowChain, Chain* columnChain, 
    SWPrefs* swPrefs) {

    SWData* swData = nwSolveGPU(rowChain, columnChain, swPrefs);

    return swData;
}
