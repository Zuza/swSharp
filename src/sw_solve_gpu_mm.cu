#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

extern "C" {
    #include "sw_solve_gpu_mm.h"

    #include "sw_prefs.h"
    #include "sw_data.h"
    #include "chain.h"
    #include "matcher.h"
    #include "utils.h"
}

#define ALPHA 5
#define GPU_CNST CONF_SLV_MM

#include "templates/mm.template.h"
#include "templates/generic.h"

#include "templates/sw_solve.template"

extern SWData* swSolveGPUMM(Chain* rowChain, Chain* columnChain, 
    SWPrefs* swPrefs) {

    SWData* swData = swSolveGPU(rowChain, columnChain, swPrefs);

    return swData;
}
