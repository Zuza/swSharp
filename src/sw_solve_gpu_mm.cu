/*
SW# - CUDA parallelized Smith Waterman with applying Hirschberg's algorithm
Copyright (C) 2011 Matija Korpar

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

Contact the author by mkorpar@gmail.com.
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

extern "C" {
    #include "sw_solve_gpu_mm.h"

    #include "sw_prefs.h"
    #include "sw_data.h"
    #include "chain.h"
    #include "matcher.h"
    #include "utils.h"
}

#define ALPHA 5
#define GPU_CNST CONF_SLV_MM

#include "templates/mm.template.h"
#include "templates/generic.h"

#include "templates/sw_solve.template"

extern SWData* swSolveGPUMM(Chain* rowChain, Chain* columnChain, 
    SWPrefs* swPrefs) {

    SWData* swData = swSolveGPU(rowChain, columnChain, swPrefs);

    return swData;
}
