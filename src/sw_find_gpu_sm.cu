#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

extern "C" {
    #include "sw_find_gpu_sm.h"

    #include "sw_prefs.h"
    #include "sw_data.h"
    #include "chain.h"
    #include "matcher.h"
    #include "utils.h"
}

#define ALPHA 5
#define SCORE CONF_FIND_SCORE_SM
#define GPU_CNST CONF_FIND_SM

#include "templates/sm.template.h"
#include "templates/generic.h"

#include "templates/sw_find.template"

extern SWData* swFindGPUSM(Chain* rowChain, Chain* columnChain, 
    SWPrefs* swPrefs, MatcherScore score) {

    SWData* swData = swFindGPU(rowChain, columnChain, swPrefs, score);

    return swData;
}
