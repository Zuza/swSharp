#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <limits.h>

extern "C" {
    #include "sw_solve_gpu.h"
    #include "sw_reconstruct_gpu_sm.h"

    #include "sw_hirschberg_data.h"
    #include "sw_prefs.h"
    #include "sw_data.h"
    #include "chain.h"
    #include "matcher.h"
    #include "utils.h"
}

#define ALPHA 1
#define GPU_CNST CONF_RCN_SM

#include "templates/sm.template.h"
#include "templates/generic.h"

#include "templates/sw_reconstruct.template"

extern SWHirschbergData* swReconstructGPUSM(Chain* rowChain, Chain* columnChain, 
    SWPrefs* swPrefs) {

    int rows = chainGetLength(rowChain);
    int columns = chainGetLength(columnChain);

    SWHirschbergData* data = swHirschbergDataCreate();
    HirshbergParent parent = {0, 0, 0, 0, rows - 1, columns - 1};

    hirschberg(data, rowChain, columnChain, swPrefs, parent);

    return data;
}
